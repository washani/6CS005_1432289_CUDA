// Name: H.G. Manesha Washani
// Student Id: 1432289



#include <hip/hip_runtime.h>
#include <stdio.h>

/* this one of the header file. in this code need 
dynamically allocated array function. library code can use
malloc, free option */

#include <stdlib.h>

#define N 4
 
/* The __global__ indicates that this is an entry-point function running on the device. is called from host code  */
 __global__ void Matrixadd(int A[][N], int B[][N], int C[][N]){
           int g = threadIdx.x;
           int h = threadIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}

int main()
{
   int i, j =0;

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

 
     for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            C[i][j] = A[i][j] + B[i][j];
        }
    }
 
   printf("Sum of entered matrices:-\n");
 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }
 
   return 0;
}
