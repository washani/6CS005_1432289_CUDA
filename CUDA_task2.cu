// Name: H.G. Manesha Washani
// Student Id: 1432289



#include <hip/hip_runtime.h>
#include <stdio.h>

/* this one of the header file. in this code need 
dynamically allocated array function. library code can use
malloc, free option */

#include <stdlib.h>

#define N 4
 
/* The __global__ indicates that this is an entry-point function running on the device. is called from host code  */
 __global__ void Matrixadd(int A[][N], int B[][N], int C[][N]){
           int g = threadIdx.x;
           int h = threadIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}

int main()
{

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

//device copies of A, B,C
 int (*d_A)[N], (*d_B)[N], (*d_C)[N];


/* Device copies of A, B and C allovated space for device aopies of A, B and C. in lecture CUDA part 1 explanation have allocate memory on the device. */

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

/* Copy input to device. the memory areas may not overlap calling cuda Memcpy()*/
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

//Launch add() kernel on GPU
  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  Matrixadd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  // Copy result back to the host
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int g, h; printf("C = \n");
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
            printf("%d ", C[g][h]);
        }
        printf("\n");
    }

// This is cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");
 
   return 0;
}
