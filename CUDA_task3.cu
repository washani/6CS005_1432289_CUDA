// Name: H.G. Manesha Washani
// Student Id: 1432289



#include <hip/hip_runtime.h>
#include <stdio.h>

/* this one of the header file. in this code need 
dynamically allocated array function. library code can use
malloc, free option */

#include <stdlib.h>

#define N 4
 
/* The __global__ indicates that this is an entry-point function running on the device. is called from host code  */
 __global__ void Matrixadd(int A[][N], int B[][N], int C[][N]){
           int g = threadIdx.x;
           int h = threadIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}

/* This is randam function of assessment gave*/

void randmatfunc(int newmat[N][N]){ //int change to void mode and added newmate parameter to the function
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
    }
}
/*Remove matrix numbers and, insert function to A and B matrix and create number automatically */
int main()
{

  int A[N][N];
  randmatfunc(A);

  int B[N][N];
  randmatfunc(B);

  int C[N][N];

//device copies of A, B,C
 int (*d_A)[N], (*d_B)[N], (*d_C)[N];


/* Device copies of A, B and C allovated space for device aopies of A, B and C. in lecture CUDA part 1 explanation have allocate memory on the device. */

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

/* Copy input to device. the memory areas may not overlap calling cuda Memcpy()*/
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

//Launch add() kernel on GPU
  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  Matrixadd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  // Copy result back to the host
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int g, h; printf("C = \n");
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
            printf("%d ", C[g][h]);
        }
        printf("\n");
    }

// This is cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");
 
   return 0;
}

